#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#ifdef USE_ROCM
#include <ATen/cuda/detail/BFloat16Utils.cuh>
#include <hip/hip_bfloat16.h>
#endif

namespace at {
namespace cuda {
namespace detail {

__global__ void out_of_place_fp32_to_bf16_kernel(float* in, uint16_t* out, int nElements)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  for(int i=id; i<nElements; i+=blockDim.x*gridDim.x)
  {
    uint32_t v = reinterpret_cast<uint32_t&>(in[i]);
    out[i] = v>>16;
  }
}
#ifdef USE_ROCM
void out_of_place_fp32_to_bf16(void* in, void* out, int nElements, hipStream_t stream)
{
  int blocks = std::min(1024, (nElements+255)/256);
  int threads = 256;
  hipLaunchKernelGGL(out_of_place_fp32_to_bf16_kernel, dim3(blocks, 1, 1), dim3(threads, 1, 1), 0, stream, (float*)in, (uint16_t*)out, nElements);
}
#endif
}
}
}

